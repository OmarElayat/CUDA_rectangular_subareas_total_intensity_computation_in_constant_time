
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <time.h>

using namespace std;

#define width_N_M 1200
#define height_N 1300
#define height_M 1500
#define MAX_FLOAT_NUM 100.0

#define TILE_DIM 32.0
#define TILE_DIM_INT int(TILE_DIM)

__global__ void MatrixMultiplykernel(float* M, float* N, float* P, int Width, int heightN, int heightM)
{
 // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
 // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x + threadIdx.x;
    if (Col < heightN && Row < heightM)
    {
    /* code */
        float Pvalue = 0;
 // Each thread computes one element of the block sub-matrix
       for (int k = 0; k < Width; ++k)
            Pvalue += M[Row*Width+k] * N[k*heightN+Col];

        P[Row*heightN+Col] = Pvalue;
    }
}

void MatrixMulOnHost (float * M, float * N, float * P, int width, int heightN, int heightM) {
    for (int i = 0; i < heightM; ++i){
        for (int j = 0; j < heightN; ++j) {
            float sum = 0;
            for (int k = 0; k < width; ++k) {
                float a = M[i * width + k];
                float b = N[k * heightN + j];
                sum += a * b;
            }
            P[i*heightN+j]=sum;
        }
    }
}

void MatrixMultiplyOnDevice (float * M, float * N, float * P, int width, int heightN, int heightM , double & gflops)  {

    int sizeN = width * heightN * sizeof(float);
    int sizeM = width * heightM * sizeof(float);
    int sizeout = heightN * heightM * sizeof(float);

    float *d_Pout, *d_PinM , *d_PinN;

    hipError_t err1 = hipMalloc((void**) &d_PinM, sizeM);

    if (err1!= hipSuccess) {
        printf("%s in %s at line %d\n",
        hipGetErrorString(err1), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy( d_PinM, M,  sizeM, hipMemcpyHostToDevice);

    hipError_t err2 = hipMalloc((void**) &d_PinN, sizeN);
    if (err2!= hipSuccess) {
        printf("%s in %s at line %d\n",
        hipGetErrorString(err2), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }


    hipMemcpy( d_PinN, N,  sizeN, hipMemcpyHostToDevice);

    hipError_t err3 = hipMalloc((void**) &d_Pout, sizeout);
    if (err3!= hipSuccess) {
        printf("%s in %s at line %d\n",
        hipGetErrorString(err3), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }


    dim3 dimGrid (ceil(heightN/32.0), ceil(heightM/32.0),1);
    dim3 dimBlock (32,32,1);

    double op_num = (double)((double)heightM * (double)heightN * (2 * (double)width - 1 ));
    clock_t start_dev = clock();
    MatrixMultiplykernel<<<dimGrid, dimBlock>>>(d_PinM,  d_PinN, d_Pout,width, heightN , heightM);
    hipDeviceSynchronize();
    clock_t stop_dev = clock();
    double time_spent_device = (double)(stop_dev - start_dev) / CLOCKS_PER_SEC;
    gflops = op_num / time_spent_device;

    hipMemcpy(P, d_Pout, sizeout, hipMemcpyDeviceToHost);

    hipFree(d_PinN); 
    hipFree(d_PinM);
    hipFree(d_Pout);
}

int main (){

    float *Pin_N,*Pin_M,*Pout, *pout_device;
    Pin_N = (float*) malloc(width_N_M * height_N * sizeof(float));
    Pin_M = (float*) malloc(width_N_M * height_M * sizeof(float));
    Pout = (float*) malloc(height_M * height_N * sizeof(float));
    pout_device = (float*) malloc(height_M * height_N * sizeof(float));
    for (int i = 0; i < height_M * width_N_M; i++)
    {
       // Pin_M[i] =  (float) (rand() %((float)100.0));
      Pin_M[i] = static_cast <float> (rand()) / 
                 (static_cast <float> (RAND_MAX/MAX_FLOAT_NUM));
    }
    for (int i = 0; i < height_N * width_N_M; i++)
    {
       // Pin_N[i] =  (float) rand()%((float)100.0) ;
        Pin_N[i] = static_cast <float> (rand()) / 
                 (static_cast <float> (RAND_MAX/MAX_FLOAT_NUM));
    }

    double op_num = (double)((double)height_M * (double)height_N * (2 * (double)width_N_M - 1 ));
    //           HOST PART
    clock_t start = clock();
    MatrixMulOnHost(Pin_M, Pin_N, Pout, width_N_M, height_N , height_M);
    clock_t stop = clock();
//    Mat_Multiply(Pin_N,Pin_M, width_N_M, height);
    double time_spent_host = (double)(stop - start) / CLOCKS_PER_SEC;
    double GFLOPS_host = op_num / time_spent_host;


    //         device Part 
     double GFLOPS_device_kernel;
    clock_t start_dev = clock();
    MatrixMultiplyOnDevice(Pin_M, Pin_N, pout_device, width_N_M, height_N , height_M, GFLOPS_device_kernel);
    hipDeviceSynchronize();
    clock_t stop_dev = clock();
//    Mat_Multiply(Pin_N,Pin_M, width_N_M, height);
    double time_spent_device = (double)(stop_dev - start_dev) / CLOCKS_PER_SEC;
    double GFLOPS_device_wrapper = op_num / time_spent_device;


    cout << "vectors size:  M " << height_M << " * " << width_N_M << endl;
    cout << "vectors size:  N " << width_N_M << " * " <<  height_N<< endl;
    cout << "Result vector size: " << height_M << " * " <<  height_N<< endl;

    // for (int i = 0; i < height_M; i++)
    // {
    //     for (int j = 0; j < width_N_M; j++)
    //     {
    //         cout << "Pin_M[" << i << "]" << "[" << j << "] = "  <<  (float) Pin_M[i*width_N_M + j] << "    " ;
    //     }
    //     cout <<endl;
    // }
    // for (int i = 0; i < width_N_M; i++)
    // {
    //     for (int j = 0; j < height_N; j++)
    //     {
    //         cout << "Pin_N[" << i << "]" << "[" << j << "] = "  <<  (float) Pin_N[i*height_N + j] << "    " ;
    //     }
    //     cout <<endl;
    // }
    // cout <<endl;

    //                      logical error checking
    for (int i = 0; i < height_M; i++)
    {
        for (int j = 0; j < height_N; j++)
        {
            if ( pout_device[i*height_N + j] - Pout[i*height_N + j] > (float) 10.1)
             {
                 cout << "Logical Error in values "<<endl;
                 cout << "pout device = " << pout_device[i*height_N + j] << "   pout = " << Pout[i*height_N + j] << endl;
                 exit(EXIT_FAILURE);
             } 
        }
    }
    cout <<endl;
    cout << " GFLOPS on host = " << GFLOPS_host/10e9 << endl;
    cout << " GFLOPS on device (wraaper) = " << GFLOPS_device_wrapper/10e9 << endl;
    cout << " GFLOPS on device (kernel) = " << GFLOPS_device_kernel/10e9 << endl;
    cout << " Speedup device(wrapper) vs host =  " << GFLOPS_device_wrapper/GFLOPS_host << endl;
    cout << " Speedup device(kernel) vs host =  " << GFLOPS_device_kernel/GFLOPS_host << endl;


    return 0;
}